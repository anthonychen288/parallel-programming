/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
//void init_line(void);
//void update (void);
void printfinal (void);
void InitUpdateOnDevice(void );
__global__ void updateKer(float* , int ,int );

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */


/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/


/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/

void InitUpdateOnDevice()
{
	float *vald;
	long long size = tpoints * sizeof(float);
	
	// allocate mem space and write to device
	hipMalloc(&vald, size);
	//cudaMemcpy(vald, values, size, cudaMemcpyHostToDevice);

	// kernel function
    int dimBlock = 20;
    int dimGrid = tpoints / dimBlock;
	updateKer<<<dimGrid, dimBlock>>>(vald, nsteps, tpoints);

	// read values from device
	hipMemcpy(values, vald, size, hipMemcpyDeviceToHost);
	// free mem space
	hipFree(vald);
}

__global__ void updateKer(float* vald, int nsteps, int tpoints)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1, n;
	float newval, oldval, val;
	__shared__ float fac, k, tmp;
	__shared__ float dtime, c, dx;
	__shared__ float tau, sqtau;

	fac = 2.0 * PI; k = i - 1; tmp = tpoints - 1;
    
    val = sin(fac * k / tmp);

    dtime = 0.3;c = 1.0;dx = 1.0;
    tau = (c * dtime / dx); sqtau = tau * tau;
	//val = vald[i];
    oldval = val;


	for(n = 1;n<=nsteps;n++)
	{
		if((i == 1) || (i == tpoints))
			newval = 0.0;
		else{
			newval = (2.0 * val) - oldval + (sqtau *  (-2.0)*val);
        }
		oldval = val;
		val = newval;
	}

    vald[i] = val;

}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}


/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
	printf("Initializing points on the line...\n");
	//init_line();
	printf("Updating all points for all time steps...\n");
	InitUpdateOnDevice();
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	
	return 0;
}